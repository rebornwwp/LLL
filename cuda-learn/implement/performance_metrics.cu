#include "hip/hip_runtime.h"
#include <stdio.h>

// https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/

// metric:
//   bandwidth throughtput: GB/s = BWEffective = (RB + WB) / (t * 10^9)
//   computational throughtput: GFLOP/s  Giga-FLoating-point OPerations per
//   second = total_operation_number / (t *10^9)
//       在saxpy中使用了一次加法、一次乘法， 所以操作总次数为 2*N/t/10^9

__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a * x[i] + y[i];
  }
}

// kernel launch time
void measure_kernel_launch_time() {
  //   hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  //   hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  //   t1 = myCPUTimer();
  //   saxpy<<<(N + 255) / 256, 256>>>(N, 2.0, d_x, d_y);
  //   hipDeviceSynchronize();
  //   t2 = myCPUTimer();

  //   hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
}

// kernel execution time
void measure_kernel_execution_time() {
  int N = 20 * (1 << 20);
  float *x, *y, *d_x, *d_y;

  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  // Perform SAXPY on 1M elements
  saxpy<<<(N + 511) / 512, 512>>>(N, 2.0f, d_x, d_y);
  hipEventRecord(stop);

  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i] - 4.0f));
  }

  printf("Max error: %f\n", maxError);
  printf("Effective Bandwidth (GB/s): %f\n", N * 4 * 3 / milliseconds / 1e6);
  // N*4 is the number of bytes transferred per array read or write
  // float is 4 bytes
  // 3 represents the reading of x and the reading and writing of y

  printf("computational throughtput (GFLOP/s): %f\n",
         N * 2 / milliseconds / 1e6);
}

int main() {
  measure_kernel_execution_time();
  return 0;
}