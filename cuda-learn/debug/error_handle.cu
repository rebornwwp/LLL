
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

#define CUDACHECK(err)                     \
  do {                                     \
    cuda_check((err), __FILE__, __LINE__); \
  } while (false)
inline void cuda_check(hipError_t error_code, const char* file, int line) {
  if (error_code != hipSuccess) {
    fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code,
            hipGetErrorString(error_code), file, line);
    fflush(stderr);
    exit(error_code);
  }
}

#define CHECK_LAST_CUDA_ERROR()   \
  do {                            \
    checkLast(__FILE__, __LINE__) \
  } while (false)
inline void checkLast(const char* const file, const int line) {
  hipError_t const err{hipGetLastError()};
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << std::endl;
    // We don't exit when we encounter CUDA errors in this example.
    // std::exit(EXIT_FAILURE);
  }
}

int main() {
  hipError_t err1 = hipPeekAtLastError();
  hipError_t errSync = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
  CUDACHECK(err1);
  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

  printf("%s\n", hipGetErrorName(err1));
}