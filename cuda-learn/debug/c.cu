
#include <cstring>
#include <iostream>
#include <string>
#include <vector>

#include "hip/hip_runtime.h"


#define cudaCheckError(e)                                      \
  {                                                            \
    if (e != hipSuccess) {                                    \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
             hipGetErrorString(e));                           \
      exit(0);                                                 \
    }                                                          \
  }

#define WIDTH 7024
#define HEIGHT 2024
#define DEPTH 4

// CUDA 内核函数：对 2D 数组中的每个元素执行加法操作
__global__ void gridStrideAdd2D(int *data, int *d_data, int width, int height,
                                int depth) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int idz = blockIdx.z * blockDim.z + threadIdx.z;

  int strideX = blockDim.x * gridDim.x;
  int strideY = blockDim.y * gridDim.y;
  int strideZ = blockDim.z * gridDim.z;

  for (int z = idz; z < depth; z += strideZ) {
    for (int y = idy; y < height; y += strideY) {
      for (int x = idx; x < width; x += strideX) {
        int index = z * width * height + y * width + x;
        data[index] = d_data[index];
      }
    }
  }
}

int main() {
  // 1. 初始化主机数据
  int size = WIDTH * HEIGHT * DEPTH * sizeof(int);
  int *h_data = new int[WIDTH * HEIGHT * DEPTH];
  int *result_data = new int[WIDTH * HEIGHT * DEPTH];
  for (int i = 0; i < WIDTH * HEIGHT * DEPTH; i++) {
    h_data[i] = 1;
  }

  // 2. 分配设备内存并将数据拷贝到设备
  int *d_data;
  hipMalloc((void **)&d_data, size);
  hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

  int *r_data;
  hipMalloc((void **)&r_data, size);
  // 3. 启动内核，使用 Grid-Stride Loop 处理 2D 数据
  dim3 blockSize(8, 8, 8);
  dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x,
                (HEIGHT + blockSize.y - 1) / blockSize.y,
                (DEPTH + blockSize.z - 1) / blockSize.z);
  gridStrideAdd2D<<<gridSize, blockSize>>>(r_data, d_data, WIDTH, HEIGHT,
                                           DEPTH);

  cudaCheckError(hipGetLastError());
  // 4. 拷贝结果回主机
  hipMemcpy(result_data, r_data, size, hipMemcpyDeviceToHost);

  // 5. 打印部分结果

  for (int index = 0; index < WIDTH * HEIGHT * DEPTH; index++) {
    if (h_data[index] != result_data[index]) {
      printf("index: %d, value1 %d, %d\n", index, h_data[index],
             result_data[index]);
    }
  }

  // 6. 释放内存
  hipFree(d_data);
  hipFree(r_data);
  delete[] h_data;
  delete[] result_data;

  return 0;
}
