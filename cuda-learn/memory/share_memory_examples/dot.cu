#include <hip/hip_runtime.h>

const int N = 33 * 1024;
const int threadsPerBlock = 256;

__global__ void inner_dot(float *a, float *b, float *c) {
  __shared__ float cache[threadsPerBlock];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;
  float temp = 0;
  while (tid < N) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cacheIndex] = temp;
  __syncthreads();

  int i = blockDim.x / 2;
  while (i != 0) {
    if (cacheIndex < i) {
      cache[cacheIndex] += cache[cacheIndex + i];
    }
    __syncthreads();  // 此时全部threads都会在这里pending
    i /= 2;
  }

  if (cacheIndex == 0) {
    c[blockIdx.x] = cache[0];
  }
}

__global__ void inner_dot_incorrect(float *a, float *b, float *c) {
  __shared__ float cache[threadsPerBlock];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;
  float temp = 0;
  while (tid < N) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cacheIndex] = temp;
  __syncthreads();

  int i = blockDim.x / 2;
  while (i != 0) {
    if (cacheIndex < i) {
      cache[cacheIndex] += cache[cacheIndex + i];
      __syncthreads();  // 这里一些threads能走到这里， 有些threads
                        // 走不到这个函数， 导致程序将会一直在等待中
    }
    i /= 2;
  }

  if (cacheIndex == 0) {
    c[blockIdx.x] = cache[0];
  }
}