
#include <hip/hip_runtime.h>

#define TPB 64
#define RAD 1

__global__ void ddKernel_global(float *d_out, const float *d_in, int size,
                                float h) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= size) return;

  d_out[i] = (d_in[i - 1] - 2.f * d_in[i] + d_in[i + 1]) / (h * h);
}

__global__ void ddKernel(float *d_out, const float *d_in, int size, float h) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= size) return;

  const int s_idx = threadIdx.x + RAD;
  extern __shared__ float s_in[];

  s_in[s_idx] = d_in[i];

  if (threadIdx.x < RAD) {
    s_in[s_idx - RAD] = d_in[i - RAD];
    s_in[s_idx + blockDim.x] = d_in[i + blockDim.x];
  }

  __syncthreads();

  d_out[i] = (s_in[s_idx - 1] - 2.f * s_in[s_idx] + s_in[s_idx + 1]) / (h * h);
}

void ddParallel(float *out, const float *in, int n, float h) {
  float *d_in = 0, *d_out = 0;
  hipMalloc(&d_in, n * sizeof(float));
  hipMalloc(&d_out, n * sizeof(float));
  hipMemcpy(d_in, in, n * sizeof(float), hipMemcpyHostToDevice);

  // set shared memory size in bytes
  const size_t smemSize = (TPB + 2 * RAD) * sizeof(float);

  ddKernel<<<(n + TPB - 1) / TPB, TPB, smemSize>>>(d_out, d_in, n, h);

  hipMemcpy(out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}