
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

// https://www.cnblogs.com/1024incn/p/4605502.html
// https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/

// Shared memory is allocated per thread block, so all threads in the block have
// access to the same shared memory. alternative is that share memory supports
// efficient sharing of information between threads in a block

// On devices of compute capability 2.x and 3.x, each multiprocessor has 64KB of
// on-chip memory that can be partitioned between L1 cache and shared memory.

// For devices of compute capability 2.x, there are two settings, 48KB shared
// memory / 16KB L1 cache, and 16KB shared memory / 48KB L1 cache. default 48KB
// shared memory

// setting level
// all kernels cudaDeviceSetCacheConfig()
// per-kernel cudaFuncSetCacheConfig()

// capability 3.x allow a third setting of 32KB shared memory / 32KB L1 cache
// which can be obtained using the option cudaFuncCachePreferEqual.

// data race problem
// __syncthreads() waits until all threads in the thread block have reached this
// point. Is used to coordinate the communication between threads in the same
// block.

// 在SIMT parallelism里，每个thread只能访问自己的kernel variable，
// 但是不能访问其他thread的东西 通过使用shared
// memory可将使得一个thread里面可以访问另外一个thread放置的信息

// ********************************** BASIC *****************************

// allocate the shared array with a fixed size
__global__ void staticReverse(int *d, int n) {
  // the shared memory array size is known at compile time.
  // size is 64
  __shared__ int s[64];
  int t = threadIdx.x;
  int tr = n - t - 1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

// allocate the shared array dynamically
__global__ void dynamicReverse(int *d, int n) {
  // dynamic shared memory
  extern __shared__ int s[];
  int t = threadIdx.x;
  int tr = n - t - 1;
  s[t] = d[t];
  __syncthreads();
  d[t] = d[tr];
}

void basicMain() {
  constexpr int N = 64;
  int a[N], r[N], d[N];

  for (int i = 0; i < N; i++) {
    a[i] = i;
    r[i] = N - i - 1;
    d[i] = 0;
  }
  printf("shared memory\n");

  int *d_d;
  hipMalloc(&d_d, N * sizeof(int));
  // run version with static shared memory
  hipMemcpy(d_d, a, N * sizeof(int), hipMemcpyHostToDevice);
  staticReverse<<<1, N>>>(d_d, N);
  hipMemcpy(d, d_d, N * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++) {
    if (d[i] != r[i]) {
      printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);
    }
  }

  // run dynamic shared memory version
  hipMemcpy(d_d, a, N * sizeof(int), hipMemcpyHostToDevice);
  dynamicReverse<<<1, N, N * sizeof(int)>>>(
      d_d, N);  // 这里设置shared memory大小，单位为byte
  hipMemcpy(d, d_d, N * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++) {
    if (d[i] != r[i]) {
      printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);
    }
  }
}

// ******************************* Example ************************

// https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/

// matrix transpose: https://en.wikipedia.org/wiki/Transpose

constexpr int TILE_DIM = 32;
constexpr int BLOCK_ROWS = 8;
constexpr int NUM_REPS = 100;

// Check errors and print GB/s
void postprocess(const float *ref, const float *res, int n, float ms) {
  bool passed = true;
  for (int i = 0; i < n; i++)
    if (res[i] != ref[i]) {
      printf("%d %f %f\n", i, res[i], ref[i]);
      printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  if (passed) printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms);
}

__global__ void index() {
  printf(
      "gridDim.x: %d, blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, "
      "threadIdx.y: %d\n",
      gridDim.x, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

// simple copy kernel
__global__ void copy(float *odata, float *idata) {
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    odata[(y + j) * width + x] = idata[(y + j) * width + x];
  }
}

__global__ void trans() {}

void exampleMain1(int argc, char **argv) {
  const int nx = 1024;
  const int ny = 1024;
  const int mem_size = nx * ny * sizeof(float);

  dim3 dimGrid(nx / TILE_DIM, ny / TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

  int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  checkCuda(hipGetDeviceProperties(&prop, devId));
  printf("\nDevice : %s\n", prop.name);
  printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n", nx, ny,
         TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
  printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n", dimGrid.x, dimGrid.y,
         dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

  checkCuda(hipSetDevice(devId));

  float *h_idata = (float *)malloc(mem_size);
  float *h_cdata = (float *)malloc(mem_size);
  float *h_tdata = (float *)malloc(mem_size);
  float *gold = (float *)malloc(mem_size);

  float *d_idata, *d_cdata, *d_tdata;
  checkCuda(hipMalloc(&d_idata, mem_size));
  checkCuda(hipMalloc(&d_cdata, mem_size));
  checkCuda(hipMalloc(&d_tdata, mem_size));

  // check parameters and calculate execution configuration
  if (nx % TILE_DIM || ny % TILE_DIM) {
    printf("nx and ny must be a multiple of TILE_DIM\n");
    goto error_exit;
  }

  if (TILE_DIM % BLOCK_ROWS) {
    printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
    goto error_exit;
  }

  // host
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++) h_idata[j * nx + i] = j * nx + i;

  // correct result for error checking
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++) gold[j * nx + i] = h_idata[i * nx + j];

  // device
  checkCuda(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

  // events for timing
  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  float ms;

  // ------------
  // time kernels
  // ------------
  printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");

  // index
  index<<<dimGrid, dimBlock>>>();

  // ----
  // copy
  // ----
  printf("%25s", "copy");
  checkCuda(hipMemset(d_cdata, 0, mem_size));
  // warm up
  copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < NUM_REPS; i++) {
    copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  checkCuda(hipMemcpy(h_cdata, d_cdata, mem_size, hipMemcpyDeviceToHost));
  postprocess(h_idata, h_cdata, nx * ny, ms);

error_exit:
  // cleanup
  checkCuda(hipEventDestroy(startEvent));
  checkCuda(hipEventDestroy(stopEvent));
  checkCuda(hipFree(d_tdata));
  checkCuda(hipFree(d_cdata));
  checkCuda(hipFree(d_idata));
  free(h_idata);
  free(h_tdata);
  free(h_cdata);
  free(gold);
}

int main(int argc, char **argv) {
  exampleMain1(argc, argv);
  return 0;
}
