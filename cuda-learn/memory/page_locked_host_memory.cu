#include <hip/hip_runtime.h>
#include <stdio.h>

// 普通 malloc 就是 pageable host memory.
// 使用cudaHostAlloc 就是 f page-locked host memory, sometimes called pinned
// memory, 主要特点，never page this memory out to disk
// 对于数据传输来说都是使用DMA进行数据的拷贝,
// 拷贝的速度主要取决于 pcie 的transfer的速度与system front-side bus speeds
// 但是使用pageable host memmory的时候 copy 会执行两次，
// 第一次:from a pageable system buffer to a page-locked “staging” buffer
// 第二次:then from the page-locked system buffer to the GPU.
//
// 使用cudahostAlloc接口申请的内存已经是page-locked， 就只用走第二步

#define CUDACHECK(err)                     \
  do {                                     \
    cuda_check((err), __FILE__, __LINE__); \
  } while (false)
inline void cuda_check(hipError_t error_code, const char *file, int line) {
  if (error_code != hipSuccess) {
    fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code,
            hipGetErrorString(error_code), file, line);
    fflush(stderr);
    exit(error_code);
  }
}

// 计算 mallloc -> devce
float cuda_malloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsedTime;

  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  a = (int *)malloc(size * sizeof(*a));

  CUDACHECK(hipMalloc((void **)&dev_a, size * sizeof(*dev_a)));
  CUDACHECK(hipEventRecord(start, 0));

  for (int i = 0; i < 100; i++) {
    if (up) {
      CUDACHECK(
          hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice));
    } else {
      CUDACHECK(
          hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
    }
  }

  CUDACHECK(hipEventRecord(stop, 0));
  CUDACHECK(hipEventSynchronize(stop));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, start, stop));

  free(a);
  CUDACHECK(hipFree(dev_a));
  CUDACHECK(hipEventDestroy(start));
  CUDACHECK(hipEventDestroy(stop));
  return elapsedTime;
}

float cuda_host_alloc_test(int size, bool up) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsedTime;

  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  CUDACHECK(hipHostAlloc((void **)&a, size * sizeof(*a),
                          hipHostMallocDefault));  // pinned host memory
  CUDACHECK(hipMalloc((void **)&dev_a, size * sizeof(*dev_a)));
  CUDACHECK(hipEventRecord(start, 0));

  for (int i = 0; i < 100; i++) {
    if (up) {
      CUDACHECK(
          hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice));
    } else {
      CUDACHECK(
          hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
    }
  }

  CUDACHECK(hipEventRecord(stop, 0));
  CUDACHECK(hipEventSynchronize(stop));
  CUDACHECK(hipEventElapsedTime(&elapsedTime, start, stop));

  CUDACHECK(hipHostFree(a));
  CUDACHECK(hipFree(dev_a));
  CUDACHECK(hipEventDestroy(start));
  CUDACHECK(hipEventDestroy(stop));
  return elapsedTime;
}

#define SIZE (10 * 1024 * 1024)

int main(void) {
  float elapsedTime;
  float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;

  elapsedTime = cuda_malloc_test(SIZE, true);
  printf("Time using cudaMalloc: %3.1f ms\n", elapsedTime);
  printf("\t MB/s during copy up: %3.1f\n", MB / (elapsedTime / 1000));

  elapsedTime = cuda_malloc_test(SIZE, false);
  printf("Time using cudaMalloc: %3.1f ms\n", elapsedTime);
  printf("\t MB/s during copy down: %3.1f\n", MB / (elapsedTime / 1000));

  elapsedTime = cuda_host_alloc_test(SIZE, true);
  printf("Time using cudaHostAlloc: %3.1f ms\n", elapsedTime);
  printf("\t MB/s during copy up: %3.1f\n", MB / (elapsedTime / 1000));

  elapsedTime = cuda_host_alloc_test(SIZE, false);
  printf("Time using cudaHostAlloc: %3.1f ms\n", elapsedTime);
  printf("\t MB/s during copy down: %3.1f\n", MB / (elapsedTime / 1000));

  return 0;
}
