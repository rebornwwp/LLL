
#include <hip/hip_runtime.h>

constexpr int N = 256;
__constant__ float constData[N];
float data[N];

int main() {
  hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data));
  hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data));

  return 0;
}