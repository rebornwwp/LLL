
#include <hip/hip_runtime.h>
#include <stdio.h>

// 广播机制：
// 如果所有线程访问相同的常量地址，constant memory 的性能接近寄存器。
// 不适合大量数据：
// Constant memory 的大小仅 64 KB，且多线程访问不同地址会导致性能下降。
// 与其他内存的对比：
// 共享内存适用于频繁更新的小型数据。
// 全局内存适用于大量读写的数据。
// 常量内存适用于小型只读常量数据。

constexpr int N = 256;
__constant__ float constData[N];
float data[N];

__device__ float devData;
float value = 3.14;

__device__ float* devPointer;
float* ptr;

// 定义常量内存
__constant__ float constant_value;

// GPU 内核函数
__global__ void add_constant(float* matrix, int size) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  while (idx < size) {
    matrix[idx] += constant_value;  // 使用 constant memory
    idx += blockDim.x * gridDim.x;
  }
}

int main() {
  // 将数据拷贝到常量内存
  hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data));
  hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data));

  hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));

  hipMalloc(&ptr, 256 * sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));

  void* devptr = NULL;
  size_t sz;
  hipGetSymbolAddress(&devptr, HIP_SYMBOL(devPointer));
  hipGetSymbolSize(&sz, HIP_SYMBOL(devPointer));
  printf("dev pointer: %d\n", devptr);  // 8
  printf("dev pointer sz: %ld\n", sz);  // 8
  return 0;
}