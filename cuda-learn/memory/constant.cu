
#include <hip/hip_runtime.h>
#include <stdio.h>

constexpr int N = 256;
__constant__ float constData[N];
float data[N];

__device__ float devData;
float value = 3.14;

__device__ float* devPointer;
float* ptr;

int main() {
  hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data));
  hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data));

  hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));

  hipMalloc(&ptr, 256 * sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));

  void* devptr = NULL;
  size_t sz;
  hipGetSymbolAddress(&devptr, HIP_SYMBOL(devPointer));
  hipGetSymbolSize(&sz, HIP_SYMBOL(devPointer));
  printf("dev pointer: %d\n", devptr);  // 8
  printf("dev pointer sz: %ld\n", sz);   // 8
  return 0;
}