#include "hip/hip_runtime.h"
#include <math.h>

#include <iostream>

// https://developer.nvidia.com/blog/unified-memory-cuda-beginners/
// https://developer.nvidia.com/blog/unified-memory-in-cuda-6/
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=hipMemPrefetchAsync#unified-memory-programming
// TODO: read doc
// 是怎么样做数据migration的
// 替换cudamalloc的注意的点
// 编程模式到底改变了些什么？
// TODO:
// https://github.com/NVIDIA-developer-blog/code-samples/tree/master/posts/unified-memory
// TODO: https://developer.nvidia.com/blog/unified-memory-in-cuda-6/
// Example: CPU/GPU Shared Linked Lists
// Example: Eliminate Deep Copies
// UVA and UM

// This approach breaks down the barrier between host memory and
// device memory so that you can have one array that is (or at least appears to
// be) accessible from both host and device

// system requirement: a GPU with compute capability not less than 3.0 and a
// 64-bit version of either Linux

//  the Pascal GPU architecture is the first with hardware support for virtual
//  memory page faulting and migration, via its Page Migration Engine

// benefit:
//   Simpler Programming and Memory Model
//   Performance Through Data Locality

// An important point is that a carefully tuned CUDA program that uses streams
// and hipMemcpyAsync to efficiently overlap execution with data transfers may
// very well perform better than a CUDA program that only uses Unified Memory.

__global__ void init(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
}

__global__ void add(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    y[i] = x[i] + y[i];
  }
}

void main_init_in_host() {
  int N = 1 << 20;
  float *x, *y;

  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x, y);

  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
}

void main_init_in_kernel() {
  int N = 1 << 20;
  float *x, *y;

  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  // initialize x and y arrays on the device
  init<<<numBlocks, blockSize>>>(N, x, y);
  hipDeviceSynchronize();

  // Launch kernel on 1M elements on the GPU
  add<<<numBlocks, blockSize>>>(N, x, y);

  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
}

void main_sync_by_user() {
  int device = -1;
  hipGetDevice(&device);

  int N = 1 << 20;
  float *x, *y;

  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // prefetch to device
  hipMemPrefetchAsync(x, N * sizeof(float), device, NULL);
  hipMemPrefetchAsync(y, N * sizeof(float), device, NULL);

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  // Launch kernel on 1M elements on the GPU
  add<<<numBlocks, blockSize>>>(N, x, y);

  // prefetch to host
  hipMemPrefetchAsync(x, N * sizeof(float), device, NULL);
  hipMemPrefetchAsync(y, N * sizeof(float), device, NULL);

  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
}

int main() {
  main_init_in_host();
  main_init_in_kernel();
  main_sync_by_user();
  return 0;
}