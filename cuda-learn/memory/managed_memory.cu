
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__managed__ unsigned int x;

// Print a managed variable
__global__ void PrintFoo() { printf("mFoo GPU: %d\n", x); }

int main() {
  x = 10;
  return 0;
}