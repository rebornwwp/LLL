
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 1024

//  deal with computations where all of the threads interact to contribute to a
//  single output. Many such computations lead to a pattern known as reduction

// reduce sum to one output value

#define TPB 64
#define ATOMIC 1  // 0 for non-atomic addition

__global__ void dotKernel(int *d_res, const int *d_a, const int *d_b, int n) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx >= n) return;

  const int s_idx = threadIdx.x;

  __shared__ int s_prod[TPB];
  s_prod[s_idx] = d_a[idx] * d_b[idx];
  __syncthreads();

  if (s_idx == 0) {
    int blockSum = 0;
    for (int j = 0; j < blockDim.x; ++j) {
      blockSum += s_prod[j];
    }
    printf("Block_%d, blockSum = %d\n", blockIdx.x, blockSum);
    // Try each of two versions of adding to the accumulator
    if (ATOMIC) {
      atomicAdd(d_res, blockSum);
    } else {
      *d_res += blockSum;
    }
  }
}

void dotLauncher(int *res, const int *a, const int *b, int n) {
  int *d_res;
  int *d_a = 0;
  int *d_b = 0;

  hipMalloc(&d_res, sizeof(int));
  hipMalloc(&d_a, n * sizeof(int));
  hipMalloc(&d_b, n * sizeof(int));

  hipMemset(d_res, 0, sizeof(int));
  hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

  dotKernel<<<(n + TPB - 1) / TPB, TPB>>>(d_res, d_a, d_b, n);
  hipMemcpy(res, d_res, sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_res);
  hipFree(d_a);
  hipFree(d_b);
}

// 还可以通过shared memory 来优化这个函数
__global__ void histo_kernel(unsigned char *buffer, long size,
                             unsigned int *histo) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  while (i < size) {
    atomicAdd(&(histo[buffer[i]]), 1);
    i += stride;
  }
}

int main() {
  int cpu_res = 0;
  int gpu_res = 0;
  int *a = (int *)malloc(N * sizeof(int));
  int *b = (int *)malloc(N * sizeof(int));

  // Initialize input arrays
  for (int i = 0; i < N; ++i) {
    a[i] = 1;
    b[i] = 1;
  }

  for (int i = 0; i < N; ++i) {
    cpu_res += a[i] * b[i];
  }
  printf("cpu result = %d\n", cpu_res);

  dotLauncher(&gpu_res, a, b, N);
  printf("gpu result = %d\n", gpu_res);

  free(a);
  free(b);
  return 0;
}