
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <vector>

// error checking macro
#define cudaCheckErrors(msg)                                  \
  do {                                                        \
    hipError_t __err = hipGetLastError();                   \
    if (__err != hipSuccess) {                               \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg, \
              hipGetErrorString(__err), __FILE__, __LINE__); \
      fprintf(stderr, "*** FAILED - ABORTING\n");             \
      exit(1);                                                \
    }                                                         \
  } while (0)

const int DSIZE = 4096;
const int block_size = 256;  // CUDA maximum is 1024
// vector add kernel: C = A + B
__global__ void vadd(const float *A, const float *B, float *C, int ds) {
  int idx =
      (blockIdx.x * blockDim.x) +
      threadIdx.x;  // create typical 1D thread index from built-in variables
  if (idx < ds)     // do the vector (element) add here
  {
    C[idx] = A[idx] + B[idx];
  }
}
using namespace std;

int main() {
  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
  h_A = new float[DSIZE];  // allocate space for vectors in host memory
  h_B = new float[DSIZE];
  h_C = new float[DSIZE];
  for (int i = 0; i < DSIZE; i++) {  // initialize vectors in host memory
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
    h_C[i] = 0;
  }
  hipMalloc(&d_A,
             DSIZE * sizeof(float));  // allocate device space for vector A
  hipMalloc(&d_B,
             DSIZE * sizeof(float));  // allocate device space for vector B
  hipMalloc(&d_C,
             DSIZE * sizeof(float));      // allocate device space for vector C
  cudaCheckErrors("hipMalloc failure");  // error checking
  // copy vector A to device:
  hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);
  // copy vector B to device:
  hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  // cuda processing sequence step 1 is complete
  vadd<<<(DSIZE + block_size - 1) / block_size, block_size>>>(d_A, d_B, d_C,
                                                              DSIZE);
  cudaCheckErrors("kernel launch failure");
  // cuda processing sequence step 2 is complete
  //  copy vector C from device to host:
  hipMemcpy(h_C, d_C, DSIZE * sizeof(float), hipMemcpyDeviceToHost);

  // cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  printf("A[0] = %f\n", h_A[0]);
  printf("B[0] = %f\n", h_B[0]);
  printf("C[0] = %f\n", h_C[0]);
  return 0;
}
