
#include <hip/hip_runtime.h>
#include <iostream>

// kernel function declation with __global__
__global__ void myKernel(void) {}

int main(void) {
  // kernel launch, run device code
  myKernel<<<1, 1>>>();
  // wait device code done
  hipDeviceSynchronize();
  printf("Hello CUDA!\n");
  return 0;
}
