
#include <hip/hip_runtime.h>

// 线程分岐（Thread Divergence） 是 CUDA 编程中性能优化的一个重要概念。它发生在
// 一个 warp 中的线程执行不同的控制流路径 时，例如在条件判断或循环中。

// CUDA 中的线程分岐
// 一个 warp 包含 32 个线程，它们被分组为一个单元并同时执行同一条指令。
// 如果 warp 内的线程在同一指令上执行不同的路径，例如 if-else 或 while
// 循环，则需要分开执行，导致性能下降。
__global__ void thread_divergence() {
  int my_var = 0;
  if (threadIdx.x % 2) {
    my_var = threadIdx.x;
  }
}