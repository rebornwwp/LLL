#include "hip/hip_runtime.h"
#include <stdio.h>

// 对于kernel函数更多的是 data parallelism
// 通过stream可以实现 task parallelism
// A CUDA stream represents a queue of GPU operations that get executed in a
// specific order

__global__ void kernel(int *a, int *b, int *c, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  while (idx < n) {
    c[idx] = a[idx] + b[idx];
    idx += stride;
  }
}
#define FULL_DATA_SIZE (1 << 20)  // 1M 元素
#define N \
  (1 << 18)  // 每次处理的大小 chunk_size, 对一个大的数据切分成几部分来计算

int main_test() {
  hipStream_t stream0, stream1;
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);
  int *host_a, *host_b, *host_c;
  int *dev_a0, *dev_a1, *dev_b0, *dev_b1, *dev_c0, *dev_c1;
  hipHostMalloc((void **)&host_a, FULL_DATA_SIZE * sizeof(*host_a));
  hipHostMalloc((void **)&host_b, FULL_DATA_SIZE * sizeof(*host_b));

  for (int i = 0; i < FULL_DATA_SIZE; i += N * 2) {
    // enqueue copies of a in stream0 and stream1
    hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice,
                    stream0);
    hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int),
                    hipMemcpyHostToDevice, stream1);
    // enqueue copies of b in stream0 and stream1
    hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice,
                    stream0);
    hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int),
                    hipMemcpyHostToDevice, stream1);
    kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0, N);
    kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1, N);

    // enqueue copies of c from device to locked memory
    hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost,
                    stream0);
    hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int),
                    hipMemcpyDeviceToHost, stream1);
  }
}

// 支持 device overlaps feature 才能发挥多stream的能力

int main(void) {
  hipDeviceProp_t prop;
  int whichDevice;
  hipGetDevice(&whichDevice);
  hipGetDeviceProperties(&prop, whichDevice);
  if (!prop.deviceOverlap) {
    printf(
        "Device will not handle overlaps, so no "
        "speed up from streams\n");
    return 0;
  }
}
