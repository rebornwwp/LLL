#include <hip/hip_runtime_api.h>
#include <stdio.h>
// 	Tesla C870	Tesla C1060	Tesla C2050	Tesla K10	Tesla
// K20

// Compute Capability	1.0	1.3	2.0	3.0	3.5

// Max Threads per Thread Block	512	512	1024	1024	1024

// Max Threads per SM	768	1024	1536	2048	2048

// Max Thread Blocks per SM	8	8	8	16	16

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    printf("  Compute capability: %d.%d\n\n", prop.major, prop.minor);
  }
}