
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__ is the qualifier for kernels (which can be called from the host
// and executed on the device).

// __host__ functions are called from the host and execute on the host.
// (This is the default qualifier and is often omitted.)

// __device__ functions are called from the device and execute on the device.
// (A function that is called from a kernel needs the __device__ qualifier.)

__global__ void call_in_host_run_in_device() {
  printf("call in host run in device\n");
}

__host__ void call_in_host_run_in_host() {
  printf("call in host run in host\n");
}

__device__ void call_in_device_run_in_device() {
  printf("call in device run in device\n");
}

__global__ void callFunctionInDeviceCode() {
  printf("call device function\n");
  call_in_device_run_in_device();
}

int main() {
  printf("#########################\n");
  call_in_host_run_in_device<<<1, 1>>>();
  hipDeviceSynchronize();
  printf("#########################\n");
  // error: a host function call cannot be configured
  // call_in_host_run_in_host<<<1, 1>>>();
  call_in_host_run_in_host();
  printf("#########################\n");
  callFunctionInDeviceCode<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}