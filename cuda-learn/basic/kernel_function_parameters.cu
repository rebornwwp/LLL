
#include <hip/hip_runtime.h>
#include <stdio.h>

// https://docs.nvidia.com/cuda/cuda-c-programming-guide/#function-parameters

// https://stackoverflow.com/questions/6499036/kernel-parameter-passing-in-cuda
// https://stackoverflow.com/questions/8302506/parameters-to-cuda-kernels

// param 在main函数中为一个host的变量，当调用device function的时候，nvcc
// compiler 将参数从host端拷贝到device端
// Passing user-defined types requires that the default copy-constructor.
// In summary pass-by-value works well for integral, floating point or other
// primitive types, and simple flat user-defined structs or class objects.

// if you are passing a pointer to a kernel, make sure it points into device
// memory.

__global__ void function_input(int param) {
  printf("input param: %d\n", param);
}

// TODO: add user defined types, need default copy-constructor
// 下面代码还不完整
struct NoMember {};
__global__ void noMemberFunc(NoMember nm) {
  printf("input no member struct: %d\n", &nm);
}

struct UserDefine {
  int a;
  int b;
};

__global__ void function_input(UserDefine ud) {}

int main() {
  constexpr int a = 10;
  function_input<<<4, 1>>>(a);
  hipDeviceSynchronize();
  return 0;
}