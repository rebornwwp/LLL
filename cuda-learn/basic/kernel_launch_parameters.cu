
#include <hip/hip_runtime.h>
#include <stdio.h>

// https://docs.nvidia.com/cuda/cuda-c-programming-guide/#execution-configuration
// https://www.informit.com/articles/article.aspx?p=2455391

//  Kernels provide dimension and index variables for each block and thread.
// Dimension variables:
// • gridDim specifies the number of blocks in the grid.
// • blockDim specifies the number of threads in each block.
// Index variables:
// • blockIdx gives the index of the block in the grid.
// • threadIdx gives the index of the thread within the block.

//  Kernels execute on the GPU and do not, in general, have access to data
//  stored on the host side that would be accessed by the CPU.

// Kernels cannot return a value, so the return type is always void

__global__ void index() {
  printf(
      "gridDim.x: %d, blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, "
      "threadIdx.y: %d\n",
      gridDim.x, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
}

__global__ void index_1D() {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  printf("1D idx: %d\n", idx);
  // 内核称为单片内核(monolithic kernel)，
  // 因为它假设一个大型线程网格可以一次性处理整个数组
  // 完全消除loop，对arrary的每个元素进行并行计算
}

// https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
__global__ void index_1D_grid_stride() {
  int n = 100;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;  // 全局索引
  int stride = blockDim.x * gridDim.x;              // 总线程数
  for (int i = tid; i < n; i += stride) {
    // operate on index i
    // 这里整体数据，切分成部分并行， 每个并行逻辑中有loop的逻辑
    // TODO: grid stride for 2D and 3D
  }
}

__global__ void index_2D() {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // if ptr is a matrix of [height][width],
  // the element of [row][col] is ptr[row * width + col]
  printf("row: %d, column: %d\n", row, col);
}

__global__ void index_3D() {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int str = blockIdx.z * blockDim.z +
            threadIdx.z;  // stratum or stack, range from [0, D-1], D is the
                          // depth of the stack
  // if ptr is a matrix of [height][width][Depth],
  // the element of [row][col][str] is ptr[row * width + col + str*width*height]
  printf("row: %d, column: %d, stratum: %d\n", row, col, str);
}

void getBoundaryValue() {
  int devNo = 0;
  hipDeviceProp_t iProp;

  hipGetDeviceProperties(&iProp, devNo);

  printf("Maximum grid size is: (");
  for (int i = 0; i < 3; i++) {
    printf("%d\t,", iProp.maxGridSize[i]);
  }
  printf(")\n");

  printf("Maximum block size is: (");
  for (int i = 0; i < 3; i++) {
    printf("%d\t,", iProp.maxThreadsDim[i]);
  }
  printf(")\n");

  printf("Max threads per block: %d\n", iProp.maxThreadsPerBlock);
  printf("Max threads per MP: %d\n", iProp.maxThreadsPerMultiProcessor);
}

int main() {
  getBoundaryValue();

  printf("####################################\n");
  printf("block 1, thread 1\n");
  index<<<1, 1>>>();
  hipDeviceSynchronize();

  printf("####################################\n");
  printf("block 2, thread 2\n");
  index<<<2, 2>>>();
  hipDeviceSynchronize();

  printf("####################################\n");
  printf("launch kernel with dim3:\n");
  int nx;  // total threads in X dimension
  int ny;  // total threads in Y dimension
  int nz;  // total threads in Z dimension
  nx = 4;
  ny = 1;
  nz = 1;
  dim3 block(2, 1, 1);
  dim3 grid(nx / block.x, ny / block.y, nz / block.z);
  printf("grid: (%d, %d, %d), block: (%d,%d,%d)\n", grid.x, grid.y, grid.z,
         block.x, block.y, block.z);
  index<<<grid, block>>>();
  hipDeviceSynchronize();

  printf("####################################\n");
  dim3 block_1D(2, 1, 1);
  dim3 grid_1D(2, 1, 1);
  printf("grid: (%d, %d, %d), block: (%d,%d,%d)\n", grid_1D.x, grid_1D.y,
         grid_1D.z, block_1D.x, block_1D.y, block_1D.z);
  index_1D<<<grid_1D, block_1D>>>();
  hipDeviceSynchronize();

  printf("####################################\n");
  dim3 block_2D(3, 3, 1);
  dim3 grid_2D(3, 3, 1);
  printf("grid: (%d, %d, %d), block: (%d,%d,%d)\n", grid_2D.x, grid_2D.y,
         grid_2D.z, block_2D.x, block_2D.y, block_2D.z);
  // matrix 2D is W columns and H rows
  // computation into 2D blocks with
  //   TX threads in the x-direction
  //   TY threads in the y-direction
  // dim3 blockSize(TX, TY);
  // int bx = (W + blockSize.x - 1)/blockSize.x ;
  // int by = (H + blockSize.y – 1)/blockSize.y ;
  // grid size:
  // dim3 gridSize = dim3(bx, by);
  index_2D<<<grid_2D, block_2D>>>();
  hipDeviceSynchronize();

  printf("####################################\n");
  dim3 block_3D(3, 3, 3);
  dim3 grid_3D(3, 3, 3);
  printf("grid: (%d, %d, %d), block: (%d,%d,%d)\n", grid_3D.x, grid_3D.y,
         grid_3D.z, block_3D.x, block_3D.y, block_3D.z);
  index_3D<<<grid_3D, block_3D>>>();
  hipDeviceSynchronize();

  printf("####################################\n");
  printf("threads size large than max:\n");
  index<<<1, 4096>>>();
  auto errName = hipGetErrorName(hipGetLastError());
  printf("get error: %s\n\n", errName);
  hipDeviceSynchronize();

  return 0;
}
