
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myKernel(void) {
  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main() {
  constexpr int block_size = 2;
  constexpr int thread_size = 2;
  myKernel<<<block_size, thread_size>>>();
  hipDeviceSynchronize();
  return 0;
}
